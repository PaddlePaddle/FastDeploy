// Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

/*
3D IoU Calculation and Rotated NMS(modified from 2D NMS written by others)
Written by Shaoshuai Shi
All Rights Reserved 2019-2020.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#define THREADS_PER_BLOCK 16
#define DIVUP(m, n) ((m) / (n) + ((m) % (n) > 0))

const int THREADS_PER_BLOCK_NMS = sizeof(int64_t) * 8;
const float EPS = 1e-8;
struct Point {
  float x, y;
  __device__ Point() {}
  __device__ Point(double _x, double _y) { x = _x, y = _y; }

  __device__ void set(float _x, float _y) {
    x = _x;
    y = _y;
  }

  __device__ Point operator+(const Point &b) const {
    return Point(x + b.x, y + b.y);
  }

  __device__ Point operator-(const Point &b) const {
    return Point(x - b.x, y - b.y);
  }
};

__device__ inline float cross(const Point &a, const Point &b) {
  return a.x * b.y - a.y * b.x;
}

__device__ inline float cross(const Point &p1, const Point &p2,
                              const Point &p0) {
  return (p1.x - p0.x) * (p2.y - p0.y) - (p2.x - p0.x) * (p1.y - p0.y);
}

__device__ int check_rect_cross(const Point &p1, const Point &p2,
                                const Point &q1, const Point &q2) {
  int ret = min(p1.x, p2.x) <= max(q1.x, q2.x) &&
            min(q1.x, q2.x) <= max(p1.x, p2.x) &&
            min(p1.y, p2.y) <= max(q1.y, q2.y) &&
            min(q1.y, q2.y) <= max(p1.y, p2.y);
  return ret;
}

__device__ inline int check_in_box2d(const float *box, const Point &p) {
  // params: (7) [x, y, z, dx, dy, dz, heading]
  const float MARGIN = 1e-2;

  float center_x = box[0], center_y = box[1];
  // rotate the point in the opposite direction of box
  float angle_cos = cos(-box[6]), angle_sin = sin(-box[6]);
  float rot_x = (p.x - center_x) * angle_cos + (p.y - center_y) * (-angle_sin);
  float rot_y = (p.x - center_x) * angle_sin + (p.y - center_y) * angle_cos;

  return (fabs(rot_x) < box[3] / 2 + MARGIN &&
          fabs(rot_y) < box[4] / 2 + MARGIN);
}

__device__ inline int intersection(const Point &p1, const Point &p0,
                                   const Point &q1, const Point &q0,
                                   Point *ans) {
  // fast exclusion
  if (check_rect_cross(p0, p1, q0, q1) == 0) return 0;

  // check cross standing
  float s1 = cross(q0, p1, p0);
  float s2 = cross(p1, q1, p0);
  float s3 = cross(p0, q1, q0);
  float s4 = cross(q1, p1, q0);

  if (!(s1 * s2 > 0 && s3 * s4 > 0)) return 0;

  // calculate intersection of two lines
  float s5 = cross(q1, p1, p0);
  if (fabs(s5 - s1) > EPS) {
    ans->x = (s5 * q0.x - s1 * q1.x) / (s5 - s1);
    ans->y = (s5 * q0.y - s1 * q1.y) / (s5 - s1);

  } else {
    float a0 = p0.y - p1.y, b0 = p1.x - p0.x, c0 = p0.x * p1.y - p1.x * p0.y;
    float a1 = q0.y - q1.y, b1 = q1.x - q0.x, c1 = q0.x * q1.y - q1.x * q0.y;
    float D = a0 * b1 - a1 * b0;

    ans->x = (b0 * c1 - b1 * c0) / D;
    ans->y = (a1 * c0 - a0 * c1) / D;
  }

  return 1;
}

__device__ inline void rotate_around_center(const Point &center,
                                            const float angle_cos,
                                            const float angle_sin, Point *p) {
  float new_x = (p->x - center.x) * angle_cos +
                (p->y - center.y) * (-angle_sin) + center.x;
  float new_y =
      (p->x - center.x) * angle_sin + (p->y - center.y) * angle_cos + center.y;
  p->set(new_x, new_y);
}

__device__ inline int point_cmp(const Point &a, const Point &b,
                                const Point &center) {
  return atan2(a.y - center.y, a.x - center.x) >
         atan2(b.y - center.y, b.x - center.x);
}

__device__ inline float box_overlap(const float *box_a, const float *box_b) {
  // params box_a: [x, y, z, dx, dy, dz, heading]
  // params box_b: [x, y, z, dx, dy, dz, heading]

  float a_angle = box_a[6], b_angle = box_b[6];
  float a_dx_half = box_a[3] / 2, b_dx_half = box_b[3] / 2,
        a_dy_half = box_a[4] / 2, b_dy_half = box_b[4] / 2;
  float a_x1 = box_a[0] - a_dx_half, a_y1 = box_a[1] - a_dy_half;
  float a_x2 = box_a[0] + a_dx_half, a_y2 = box_a[1] + a_dy_half;
  float b_x1 = box_b[0] - b_dx_half, b_y1 = box_b[1] - b_dy_half;
  float b_x2 = box_b[0] + b_dx_half, b_y2 = box_b[1] + b_dy_half;

  Point center_a(box_a[0], box_a[1]);
  Point center_b(box_b[0], box_b[1]);

  Point box_a_corners[5];
  box_a_corners[0].set(a_x1, a_y1);
  box_a_corners[1].set(a_x2, a_y1);
  box_a_corners[2].set(a_x2, a_y2);
  box_a_corners[3].set(a_x1, a_y2);

  Point box_b_corners[5];
  box_b_corners[0].set(b_x1, b_y1);
  box_b_corners[1].set(b_x2, b_y1);
  box_b_corners[2].set(b_x2, b_y2);
  box_b_corners[3].set(b_x1, b_y2);

  // get oriented corners
  float a_angle_cos = cos(a_angle), a_angle_sin = sin(a_angle);
  float b_angle_cos = cos(b_angle), b_angle_sin = sin(b_angle);

  for (int k = 0; k < 4; k++) {
    rotate_around_center(center_a, a_angle_cos, a_angle_sin, box_a_corners + k);
    rotate_around_center(center_b, b_angle_cos, b_angle_sin, box_b_corners + k);
  }

  box_a_corners[4] = box_a_corners[0];
  box_b_corners[4] = box_b_corners[0];

  // get intersection of lines
  Point cross_points[16];
  Point poly_center;
  int cnt = 0, flag = 0;

  poly_center.set(0, 0);
  for (int i = 0; i < 4; i++) {
    for (int j = 0; j < 4; j++) {
      flag = intersection(box_a_corners[i + 1], box_a_corners[i],
                          box_b_corners[j + 1], box_b_corners[j],
                          cross_points + cnt);
      if (flag) {
        poly_center = poly_center + cross_points[cnt];
        cnt++;
      }
    }
  }

  // check corners
  for (int k = 0; k < 4; k++) {
    if (check_in_box2d(box_a, box_b_corners[k])) {
      poly_center = poly_center + box_b_corners[k];
      cross_points[cnt] = box_b_corners[k];
      cnt++;
    }
    if (check_in_box2d(box_b, box_a_corners[k])) {
      poly_center = poly_center + box_a_corners[k];
      cross_points[cnt] = box_a_corners[k];
      cnt++;
    }
  }

  poly_center.x /= cnt;
  poly_center.y /= cnt;

  // sort the points of polygon
  Point temp;
  for (int j = 0; j < cnt - 1; j++) {
    for (int i = 0; i < cnt - j - 1; i++) {
      if (point_cmp(cross_points[i], cross_points[i + 1], poly_center)) {
        temp = cross_points[i];
        cross_points[i] = cross_points[i + 1];
        cross_points[i + 1] = temp;
      }
    }
  }

  // get the overlap areas
  float area = 0;
  for (int k = 0; k < cnt - 1; k++) {
    area += cross(cross_points[k] - cross_points[0],
                  cross_points[k + 1] - cross_points[0]);
  }

  return fabs(area) / 2.0;
}

__device__ inline float iou_bev(const float *box_a, const float *box_b) {
  // params box_a: [x, y, z, dx, dy, dz, heading]
  // params box_b: [x, y, z, dx, dy, dz, heading]
  float sa = box_a[3] * box_a[4];
  float sb = box_b[3] * box_b[4];
  float s_overlap = box_overlap(box_a, box_b);
  return s_overlap / fmaxf(sa + sb - s_overlap, EPS);
}

__global__ void nms_kernel(const int num_bboxes, const int num_bboxes_for_nms,
                           const float nms_overlap_thresh,
                           const int decode_bboxes_dims, const float *bboxes,
                           const int *index, const int64_t *sorted_index,
                           int64_t *mask) {
  // params: boxes (N, 7) [x, y, z, dx, dy, dz, heading]
  // params: mask (N, N/THREADS_PER_BLOCK_NMS)

  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;

  const int row_size =
      fminf(num_bboxes_for_nms - row_start * THREADS_PER_BLOCK_NMS,
            THREADS_PER_BLOCK_NMS);
  const int col_size =
      fminf(num_bboxes_for_nms - col_start * THREADS_PER_BLOCK_NMS,
            THREADS_PER_BLOCK_NMS);

  __shared__ float block_boxes[THREADS_PER_BLOCK_NMS * 7];

  if (threadIdx.x < col_size) {
    int box_idx =
        index[sorted_index[THREADS_PER_BLOCK_NMS * col_start + threadIdx.x]];
    block_boxes[threadIdx.x * 7 + 0] = bboxes[box_idx * decode_bboxes_dims];
    block_boxes[threadIdx.x * 7 + 1] = bboxes[box_idx * decode_bboxes_dims + 1];
    block_boxes[threadIdx.x * 7 + 2] = bboxes[box_idx * decode_bboxes_dims + 2];
    block_boxes[threadIdx.x * 7 + 3] = bboxes[box_idx * decode_bboxes_dims + 4];
    block_boxes[threadIdx.x * 7 + 4] = bboxes[box_idx * decode_bboxes_dims + 3];
    block_boxes[threadIdx.x * 7 + 5] = bboxes[box_idx * decode_bboxes_dims + 5];
    block_boxes[threadIdx.x * 7 + 6] =
        -bboxes[box_idx * decode_bboxes_dims + decode_bboxes_dims - 1] -
        3.141592653589793 / 2;
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = THREADS_PER_BLOCK_NMS * row_start + threadIdx.x;
    const int act_box_idx = index[sorted_index[cur_box_idx]];
    float cur_box[7];
    cur_box[0] = bboxes[act_box_idx * decode_bboxes_dims];
    cur_box[1] = bboxes[act_box_idx * decode_bboxes_dims + 1];
    cur_box[2] = bboxes[act_box_idx * decode_bboxes_dims + 2];
    cur_box[3] = bboxes[act_box_idx * decode_bboxes_dims + 4];
    cur_box[4] = bboxes[act_box_idx * decode_bboxes_dims + 3];
    cur_box[5] = bboxes[act_box_idx * decode_bboxes_dims + 5];
    cur_box[6] =
        -bboxes[act_box_idx * decode_bboxes_dims + decode_bboxes_dims - 1] -
        3.141592653589793 / 2;

    int i = 0;
    int64_t t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      if (iou_bev(cur_box, block_boxes + i * 7) > nms_overlap_thresh) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = DIVUP(num_bboxes_for_nms, THREADS_PER_BLOCK_NMS);
    mask[cur_box_idx * col_blocks + col_start] = t;
  }
}

void NmsLauncher(const hipStream_t &stream, const float *bboxes,
                 const int *index, const int64_t *sorted_index,
                 const int num_bboxes, const int num_bboxes_for_nms,
                 const float nms_overlap_thresh, const int decode_bboxes_dims,
                 int64_t *mask) {
  dim3 blocks(DIVUP(num_bboxes_for_nms, THREADS_PER_BLOCK_NMS),
              DIVUP(num_bboxes_for_nms, THREADS_PER_BLOCK_NMS));
  dim3 threads(THREADS_PER_BLOCK_NMS);
  nms_kernel<<<blocks, threads, 0, stream>>>(
      num_bboxes, num_bboxes_for_nms, nms_overlap_thresh, decode_bboxes_dims,
      bboxes, index, sorted_index, mask);
}
